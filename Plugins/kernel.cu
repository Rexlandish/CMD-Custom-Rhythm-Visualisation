#include "hip/hip_runtime.h"
extern "C" __global__ void generateKernel(float* result, int width, int height, double beat, double swirliness, double swirlDensity, double tubeSpacing, double swirlSpeed) {
    double M_PI = 3.14159265358979;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < height && j < width) {
        double _i = (double)i;
        double _j = (double)j;

        // Scrolling
        _j += beat * 8;

        double wibbleAmount = cos(beat * M_PI);

        double asidenessSin = swirliness * sin(wibbleAmount * sin(M_PI * ((swirlSpeed * beat + _i) / swirlDensity)));
        double asidenessCos = swirliness * sin(wibbleAmount * cos(M_PI * ((swirlSpeed * beat + _i) / swirlDensity)));

        double currentDensity = cos((_j / tubeSpacing) - asidenessCos);// *(asidenessSin * 0.2);

        result[i * width + j] = (float)currentDensity;
    }
}


// nvcc -ptx C:\Users\glass\Documents\Repos\ASCIIMusicVisualiserGithub\CMD-Custom-Rhythm-Visualisation\Plugins\kernel.cu -o C:\Users\glass\Documents\Repos\ASCIIMusicVisualiserGithub\CMD-Custom-Rhythm-Visualisation\Plugins\kernel.ptx